#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <cmath>

// CUDA 커널 함수
template <typename scalar_t>
__global__
void fused_kernel(
    const scalar_t* __restrict__ x, 
    const scalar_t* __restrict__ gate_proj_weights,
    const scalar_t* __restrict__ up_proj_weights, 
    const scalar_t* __restrict__ down_proj_weights,
    scalar_t* __restrict__ output, 
    int batch_size, 
    int seq_len, 
    int hidden_size, 
    int intermediate_size) {

    /* 
    the original operations
    
    output = down_proj(
        silu(gate_proj(input_tensor)) * up_proj(input_tensor)  <--- element-wise operation
    )

    */

    int batch_idx = blockIdx.x;
    int seq_idx = blockIdx.y;
    int hidden_idx = threadIdx.x;

    if (hidden_idx < hidden_size) {
        int input_idx = (batch_idx * seq_len + seq_idx) * hidden_size + hidden_idx;
        scalar_t gate_proj_val = 0.0;
        scalar_t up_proj_val = 0.0;

        for (int i = 0; i < intermediate_size; ++i) {
            gate_proj_val += x[input_idx] * gate_proj_weights[hidden_idx * intermediate_size + i];
            up_proj_val += x[input_idx] * up_proj_weights[hidden_idx * intermediate_size + i];
        }

        // SiLU activation function
        gate_proj_val = gate_proj_val / (1.0 + exp(-gate_proj_val));

        scalar_t down_proj_val = 0.0;
        for (int i = 0; i < intermediate_size; ++i) {
            down_proj_val += gate_proj_val * up_proj_val * down_proj_weights[i * hidden_size + hidden_idx];
        }

        output[input_idx] = down_proj_val;
    }
}

// Python에서 호출 가능한 인터페이스 함수
torch::Tensor fused_mlp(
    torch::Tensor x,
    torch::Tensor gate_proj_weights,
    torch::Tensor up_proj_weights,
    torch::Tensor down_proj_weights,
    int block_size) {

    // Tensor shape 확인 및 변수 설정
    auto batch_size = x.size(0);
    auto seq_len = x.size(1);
    auto hidden_size = x.size(2);
    auto intermediate_size = gate_proj_weights.size(0);

    // 출력 텐서 생성
    auto output = torch::zeros_like(x);

    // 블록 및 그리드 크기 설정
    const dim3 blocks(batch_size, seq_len);

    // dispatch 참고
    // https://github.com/pytorch/pytorch/blob/010009e6421e9ef7d4af549527594af954c3c84c/aten/src/ATen/Dispatch.h#L286
    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half,
        at::ScalarType::BFloat16,
        x.scalar_type(),
        "fused_mlp", ([&] {
            fused_kernel<scalar_t><<<blocks, block_size>>>(
                x.data_ptr<scalar_t>(),
                gate_proj_weights.data_ptr<scalar_t>(),
                up_proj_weights.data_ptr<scalar_t>(),
                down_proj_weights.data_ptr<scalar_t>(),
                output.data_ptr<scalar_t>(),
                batch_size, seq_len, hidden_size, intermediate_size
            );
    }));

    // 출력 텐서 반환
    return output;
}

// Python 모듈 초기화
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("fused_mlp", &fused_mlp, "Fused MLP kernel",
        pybind11::arg("x"),
        pybind11::arg("gate_proj_weights"),
        pybind11::arg("up_proj_weights"),
        pybind11::arg("down_proj_weights"),
        pybind11::arg("block_size")
    );
}
